#include "hip/hip_runtime.h"
#include "inference.cuh"

using namespace nvinfer1;

#include <NvOnnxParser.h>

/**
* cudaMemcpys CPU memory in inputTensor to GPU based on bindings
* Queues that tensor to be passed through model
* cudaMemcpys the result back to CPU memory
* Requires bindings, inputTensor, stream
* Modifies stream, outputTensor
*/
constexpr static auto INPUT_BINDING_NAME = "images";
constexpr static auto OUTPUT_BINDING_NAME = "output0";

Inference::Inference(std::filesystem::path const& onnxModelPath, std::string const& modelName, std::string packagePathString) : mPackagePath{std::move(packagePathString)} {
    mModelPath = onnxModelPath.string();

    // Create the engine object from either the file or from onnx file
    mEngine = std::unique_ptr<ICudaEngine>{createCudaEngine(onnxModelPath, modelName)};
    if (!mEngine) throw std::runtime_error("Failed to create CUDA engine");

    mLogger.log(ILogger::Severity::kINFO, "Created CUDA Engine");

    // Check some assumptions about the model
    if (mEngine->getNbIOTensors() != 2) throw std::runtime_error("Invalid Binding Count");
    if (mEngine->getTensorIOMode(INPUT_BINDING_NAME) != TensorIOMode::kINPUT) throw std::runtime_error("Expected Input Binding 0 Is An Input");
    if (mEngine->getTensorIOMode(OUTPUT_BINDING_NAME) != TensorIOMode::kOUTPUT) throw std::runtime_error("Expected Input Binding Input To Be 1");

    createExecutionContext();

    prepTensors();
}

auto Inference::createCudaEngine(std::filesystem::path const& onnxModelPath, std::string const& modelName) -> ICudaEngine* {
    constexpr auto explicitBatch = 1U << static_cast<std::uint32_t>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);

    std::unique_ptr<IBuilder> builder{createInferBuilder(mLogger)};
    if (!builder) throw std::runtime_error("Failed to create Infer Builder");
    mLogger.log(ILogger::Severity::kINFO, "Created Infer Builder");

    std::unique_ptr<INetworkDefinition> network{builder->createNetworkV2(explicitBatch)};
    if (!network) throw std::runtime_error("Failed to create Network Definition");
    mLogger.log(ILogger::Severity::kINFO, "Created Network Definition");

    std::unique_ptr<nvonnxparser::IParser> parser{nvonnxparser::createParser(*network, mLogger)};
    if (!parser) throw std::runtime_error("Failed to create ONNX Parser");
    mLogger.log(ILogger::Severity::kINFO, "Created ONNX Parser");

    std::unique_ptr<IBuilderConfig> config{builder->createBuilderConfig()};
    if (!config) throw std::runtime_error("Failed to create Builder Config");
    mLogger.log(ILogger::Severity::kINFO, "Created Builder Config");

    if (!parser->parseFromFile(onnxModelPath.c_str(), static_cast<int>(ILogger::Severity::kINFO))) {
        throw std::runtime_error("Failed to parse ONNX file");
    }

    IRuntime* runtime = createInferRuntime(mLogger);

    // Define the engine file location relative to the mrover package
    std::filesystem::path packagePath{mPackagePath};
    std::filesystem::path enginePath = packagePath / "data" / std::string("tensorrt-engine-").append(modelName).append(".engine");
    // Check if engine file exists
    if (!exists(enginePath)) {
		std::cout << "Optimizing ONXX model for TensorRT. This make take a long time..." << std::endl;

        // Create the Engine from onnx file
        IHostMemory* serializedEngine = builder->buildSerializedNetwork(*network, *config);
        if (!serializedEngine) throw std::runtime_error("Failed to serialize engine");

        // Create temporary engine for serializing
        ICudaEngine* tempEng = runtime->deserializeCudaEngine(serializedEngine->data(), serializedEngine->size());
        if (!tempEng) throw std::runtime_error("Failed to create temporary engine");

        // Save Engine to File
        auto trtModelStream = tempEng->serialize();
        std::ofstream outputFileStream{enginePath, std::ios::binary};
        outputFileStream.write(static_cast<char const*>(trtModelStream->data()), trtModelStream->size());
        outputFileStream.close();

        return tempEng;
    }

    // Load engine from file
    std::ifstream inputFileStream{enginePath, std::ios::binary};
    std::stringstream engineBuffer;

    // Stream in the engine file to the buffer
    engineBuffer << inputFileStream.rdbuf();
    std::string enginePlan = engineBuffer.str();
    // Deserialize the Cuda Engine file from the buffer
    return runtime->deserializeCudaEngine(enginePlan.data(), enginePlan.size());
}

auto Inference::createExecutionContext() -> void {
    // Create Execution Context
    mContext.reset(mEngine->createExecutionContext());
    if (!mContext) throw std::runtime_error("Failed to create execution context");

    // Set up the input tensor sizing
    mContext->setInputShape(INPUT_BINDING_NAME, mEngine->getTensorShape(INPUT_BINDING_NAME));
}

auto Inference::doDetections(cv::Mat const& img) const -> void {
    // Do the forward pass on the network
    launchInference(img, mOutputTensor);
}

auto Inference::getOutputTensor() -> cv::Mat {
    return mOutputTensor;
}

auto Inference::launchInference(cv::Mat const& input, cv::Mat const& output) const -> void {
    //Assert these items have been initialized
    assert(!input.empty());
    assert(!output.empty());
    assert(input.isContinuous());
    assert(output.isContinuous());
    assert(mContext);

    // Get the binding id for the input tensor
    int inputId = getBindingInputIndex(mContext.get());

    // Memcpy the input tensor from the host to the gpu
    hipMemcpy(mBindings[inputId], input.data, input.total() * input.elemSize(), hipMemcpyHostToDevice);

    // Execute the model on the gpu
    mContext->executeV2(mBindings.data());

    // Memcpy the output tensor from the gpu to the host
    hipMemcpy(output.data, mBindings[1 - inputId], output.total() * output.elemSize(), hipMemcpyDeviceToHost);
}

auto Inference::prepTensors() -> void {
    // Assign the properties to the input and output tensors
    for (int i = 0; i < mEngine->getNbIOTensors(); i++) {
        char const* tensorName = mEngine->getIOTensorName(i);
        auto [rank, extents] = mEngine->getTensorShape(tensorName);

        // Multiply sizeof(float) by the product of the extents
        // This is essentially: element count * size of each element
        std::size_t size = 1;
		for(int32_t i = 0; i < rank; ++i){
			size *= extents[i];
		}
		std::cout << tensorName << " is getting allocated to size " << size << std::endl;

		// Create GPU memory for TensorRT to operate on
        if (hipError_t result = hipMalloc(mBindings.data() + i, size * sizeof(float)); result != hipSuccess)
            throw std::runtime_error{"Failed to allocate GPU memory: " + std::string{hipGetErrorString(result)}};
    }

    assert(mContext);
    // Create an appropriately sized output tensor
    auto const [nbDims, d] = mEngine->getTensorShape(OUTPUT_BINDING_NAME);
    for (int i = 0; i < nbDims; i++) {
        std::array<char, 512> message;
        std::snprintf(message.data(), message.size(), "Size %d %d", i, d[i]);
        mLogger.log(ILogger::Severity::kINFO, message.data());
    }

    // Create the mat wrapper around the output matrix for ease of use
    assert(nbDims == 3);
    assert(d[0] == 1);
    mOutputTensor = cv::Mat::zeros(d[1], d[2], CV_32FC1);
}

auto Inference::getBindingInputIndex(IExecutionContext const* context) -> int {
    // Returns the id for the input tensor
    return context->getEngine().getTensorIOMode(context->getEngine().getIOTensorName(0)) != TensorIOMode::kINPUT; // 0 (false) if bindingIsInput(0), 1 (true) otherwise
}
