#include "hip/hip_runtime.h"
#include "inference.cuh"
#include <ios>

using namespace nvinfer1;

#include <NvOnnxParser.h>

/**
 * cudaMemcpys CPU memory in inputTensor to GPU based on bindings
 * Queues that tensor to be passed through model
 * cudaMemcpys the result back to CPU memory
 * Requires bindings, inputTensor, stream
 * Modifies stream, outputTensor
 */

Inference::Inference(std::string modelName, std::string packagePathString) :  mModelName{std::move(modelName)}, mPackagePath{std::move(packagePathString)}{

	//Create ONNX and engine file paths
	mLogger.log(ILogger::Severity::kINFO, mModelName.c_str());
	mLogger.log(ILogger::Severity::kINFO, mPackagePath.c_str());
	mONNXModelPath = std::filesystem::path{packagePathString} / "data" / std::string(mModelName + ".onnx");
	mEngineModelPath = std::filesystem::path{packagePathString} / "data" / std::string("tensorrt-engine-" + mModelName + ".engine");;

	std::array<char, 150> message{};
	std::snprintf(message.data(), message.size(), "Reading from ONNX model at %s and creating TensorRT engine at %s", mONNXModelPath.c_str(), mEngineModelPath.c_str());
	mLogger.log(ILogger::Severity::kINFO, message.data());

	// Create the engine object from either the file or from onnx file
	mEngine = std::unique_ptr<ICudaEngine>{createCudaEngine()};
	if (!mEngine) throw std::runtime_error("Failed to create CUDA engine");

	mLogger.log(ILogger::Severity::kINFO, "Created CUDA Engine");

	// Check some assumptions about the model
	if (mEngine->getNbIOTensors() != 2) throw std::runtime_error("Invalid Binding Count");

	// Store the IO Tensor Names
	mInputTensorName = mEngine->getIOTensorName(0);
	mOutputTensorName = mEngine->getIOTensorName(1);

	if (mEngine->getTensorIOMode(mInputTensorName.c_str()) != TensorIOMode::kINPUT) throw std::runtime_error("Expected Input Binding 0 Is An Input");
	if (mEngine->getTensorIOMode(mOutputTensorName.c_str()) != TensorIOMode::kOUTPUT) throw std::runtime_error("Expected Output Binding Input To Be 1");

	// Be verbose about the input tensor size
	auto inputTensorSize = getInputTensorSize();
	std::snprintf(message.data(), message.size(), "%s Tensor's Dimensions:", mInputTensorName.c_str());
	mLogger.log(ILogger::Severity::kINFO, message.data());
	for(size_t i = 0; i < inputTensorSize.size(); ++i){
		std::snprintf(message.data(), message.size(), "Dimension: %zu Size: %zu", i, inputTensorSize[i]);
		mLogger.log(ILogger::Severity::kINFO, message.data());
	}

	// Be verbose about the input tensor size
	auto outputTensorSize = getOutputTensorSize();
	std::snprintf(message.data(), message.size(), "%s Tensor's Dimensions:", mOutputTensorName.c_str());
	mLogger.log(ILogger::Severity::kINFO, message.data());
	for(size_t i = 0; i < outputTensorSize.size(); ++i){
		std::snprintf(message.data(), message.size(), "Dimension: %zu Size: %zu", i, outputTensorSize[i]);
		mLogger.log(ILogger::Severity::kINFO, message.data());
	}

	createExecutionContext();

	prepTensors();
}

auto Inference::createCudaEngine() -> ICudaEngine* {
	mLogger.log(ILogger::Severity::kINFO, "Creating engine building tools...");
	constexpr auto explicitBatch = 1U << static_cast<std::uint32_t>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);

	IBuilder* builder = createInferBuilder(mLogger);
	if (!builder) throw std::runtime_error("Failed to create Infer Builder");
	mLogger.log(ILogger::Severity::kINFO, "Created Infer Builder");

	std::unique_ptr<INetworkDefinition> network{builder->createNetworkV2(explicitBatch)};
	if (!network) throw std::runtime_error("Failed to create Network Definition");
	mLogger.log(ILogger::Severity::kINFO, "Created Network Definition");

	std::unique_ptr<nvonnxparser::IParser> parser{nvonnxparser::createParser(*network, mLogger)};
	if (!parser) throw std::runtime_error("Failed to create ONNX Parser");
	mLogger.log(ILogger::Severity::kINFO, "Created ONNX Parser");

	std::unique_ptr<IBuilderConfig> config{builder->createBuilderConfig()};
	if (!config) throw std::runtime_error("Failed to create Builder Config");
	mLogger.log(ILogger::Severity::kINFO, "Created Builder Config");

	if (!parser->parseFromFile(mONNXModelPath.c_str(), static_cast<int>(ILogger::Severity::kINFO))) {
		throw std::runtime_error("Failed to parse ONNX file");
	}

	IRuntime* runtime = createInferRuntime(mLogger);

	// Define the engine file location relative to the mrover package
	// Check if engine file exists
	if (!exists(mEngineModelPath)) {
		std::cout << "Optimizing ONXX model for TensorRT. This make take a long time..." << std::endl;

		// Create the Engine from onnx file
		IHostMemory* serializedEngine = builder->buildSerializedNetwork(*network, *config);
		if (!serializedEngine) throw std::runtime_error("Failed to serialize engine");

		// Create temporary engine for serializing
		ICudaEngine* tempEng = runtime->deserializeCudaEngine(serializedEngine->data(), serializedEngine->size());
		if (!tempEng) throw std::runtime_error("Failed to create temporary engine");

		// Save Engine to File
		auto trtModelStream = tempEng->serialize();
		std::ofstream outputFileStream{mEngineModelPath, std::ios::binary};
		outputFileStream.write(static_cast<char const*>(trtModelStream->data()), static_cast<std::streamsize>(trtModelStream->size()));
		outputFileStream.close();

		return tempEng;
	}

	// Load engine from file
	std::ifstream inputFileStream{mEngineModelPath, std::ios::binary};
	std::stringstream engineBuffer;

	// Stream in the engine file to the buffer
	engineBuffer << inputFileStream.rdbuf();
	std::string enginePlan = engineBuffer.str();
	// Deserialize the Cuda Engine file from the buffer
	return runtime->deserializeCudaEngine(enginePlan.data(), enginePlan.size());
}

auto Inference::createExecutionContext() -> void {
	// Create Execution Context
	mContext.reset(mEngine->createExecutionContext());
	if (!mContext) throw std::runtime_error("Failed to create execution context");

	// Set up the input tensor sizing
	mContext->setInputShape(mInputTensorName.c_str(), mEngine->getTensorShape(mInputTensorName.c_str()));
}

auto Inference::doDetections(cv::Mat const& img) const -> void {
	// Do the forward pass on the network
	launchInference(img, mOutputTensor);
}

auto Inference::getOutputTensor() -> cv::Mat {
	return mOutputTensor;
}

auto Inference::launchInference(cv::Mat const& input, cv::Mat const& output) const -> void {
	//Assert these items have been initialized
	assert(!input.empty());
	assert(!output.empty());
	assert(input.isContinuous());
	assert(output.isContinuous());
	assert(mContext);

	// Get the binding id for the input tensor
	int inputId = getBindingInputIndex(mContext.get());

	// Memcpy the input tensor from the host to the gpu
	hipMemcpy(mBindings[inputId], input.data, input.total() * input.elemSize(), hipMemcpyHostToDevice);

	// Execute the model on the gpu
	mContext->executeV2(mBindings.data());

	// Memcpy the output tensor from the gpu to the host
	hipMemcpy(output.data, mBindings[1 - inputId], output.total() * output.elemSize(), hipMemcpyDeviceToHost);
}

auto Inference::prepTensors() -> void {
	// Assign the properties to the input and output tensors
	for (int i = 0; i < mEngine->getNbIOTensors(); i++) {
		char const* tensorName = mEngine->getIOTensorName(i);
		auto [rank, extents] = mEngine->getTensorShape(tensorName);

		// Multiply sizeof(float) by the product of the extents
		// This is essentially: element count * size of each element
		std::size_t size = 1;
		for(int32_t i = 0; i < rank; ++i){
			size *= extents[i];
		}

		// Create GPU memory for TensorRT to operate on
		if (hipError_t result = hipMalloc(mBindings.data() + i, size * sizeof(float)); result != hipSuccess)
			throw std::runtime_error{"Failed to allocate GPU memory: " + std::string{hipGetErrorString(result)}};
	}

	assert(mContext);
	// Create an appropriately sized output tensor
	auto const [nbDims, d] = mEngine->getTensorShape(mOutputTensorName.c_str());
	for (int i = 0; i < nbDims; i++) {
		std::array<char, 512> message{};
		std::snprintf(message.data(), message.size(), "Size %d %d", i, d[i]);
		mLogger.log(ILogger::Severity::kINFO, message.data());
	}

	// Create the mat wrapper around the output matrix for ease of use
	assert(nbDims == 3);
	assert(d[0] == 1);
	mOutputTensor = cv::Mat::zeros(d[1], d[2], CV_32FC1);
}

auto Inference::getBindingInputIndex(IExecutionContext const* context) -> int {
	// Returns the id for the input tensor
	return context->getEngine().getTensorIOMode(context->getEngine().getIOTensorName(0)) != TensorIOMode::kINPUT; // 0 (false) if bindingIsInput(0), 1 (true) otherwise
}


auto Inference::getInputTensorSize() -> std::vector<int64_t>{
	auto dims =  mEngine->getTensorShape(mInputTensorName.c_str());
	std::vector<int64_t> inputBlobSize;
	inputBlobSize.reserve(dims.nbDims);

	for(int32_t i = 0; i < dims.nbDims; ++i){
		inputBlobSize.push_back(dims.d[i]);
	}

	return inputBlobSize;
}

auto Inference::getOutputTensorSize() -> std::vector<int64_t>{
	auto dims =  mEngine->getTensorShape(mOutputTensorName.c_str());
	std::vector<int64_t> inputBlobSize;
	inputBlobSize.reserve(dims.nbDims);

	for(int32_t i = 0; i < dims.nbDims; ++i){
		inputBlobSize.push_back(dims.d[i]);
	}

	return inputBlobSize;
}
