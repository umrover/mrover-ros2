#include "hip/hip_runtime.h"
#include "logger.cuh"

namespace nvinfer1 {

    auto Logger::log(Severity severity, char const* msg) noexcept -> void {
        switch (severity) {
            case Severity::kINTERNAL_ERROR:
				std::cout << "[FATAL] " << msg << "\n";
                break;
            case Severity::kERROR:
				std::cout << "[ERROR] " << msg << "\n";
                break;
            case Severity::kWARNING:
				std::cout << "[WARN] " << msg << "\n";
                break;
            case Severity::kINFO:
				std::cout << "[INFO] " << msg << "\n";
                break;
            case Severity::kVERBOSE:
				std::cout << "[VERBOSE] " << msg << "\n";
                break;
        }
    }

} // namespace nvinfer1
