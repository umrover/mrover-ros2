#include "hip/hip_runtime.h"
// Be careful what you include in this file, it is compiled with nvcc (NVIDIA CUDA compiler)

#include "point.hpp"

#include <sensor_msgs/distortion_models.hpp>
#include <sensor_msgs/msg/point_cloud2.hpp>
#include <sensor_msgs/point_cloud2_iterator.hpp>

#include <sl/Camera.hpp>
#include <thrust/device_vector.h>

namespace mrover {

    using PointCloudGpu = thrust::device_vector<Point>;

    // Optimal for the Jetson Xavier NX - this is max threads per block and each block has a max of 2048 threads
    constexpr uint BLOCK_SIZE = 1024;

    /**
     * @brief Runs on the GPU, interleaving the XYZ and BGRA buffers into a single buffer of #Point structs.
     */
    __global__ void fillPointCloudMessageKernel(sl::float4* xyzGpuPtr, sl::uchar4* bgraGpuPtr, sl::float4* normalsGpuPtr, Point* pcGpuPtr, size_t size) {
        // This function is invoked once per element at index #i in the point cloud
        size_t const i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= size) return;

        pcGpuPtr[i].x = xyzGpuPtr[i].x;
        pcGpuPtr[i].y = xyzGpuPtr[i].y;
        pcGpuPtr[i].z = xyzGpuPtr[i].z;
        pcGpuPtr[i].b = bgraGpuPtr[i].r;
        pcGpuPtr[i].g = bgraGpuPtr[i].g;
        pcGpuPtr[i].r = bgraGpuPtr[i].b;
        pcGpuPtr[i].a = bgraGpuPtr[i].a;
        // pcGpuPtr[i].normal_x = normalsGpuPtr[i].x;
        // pcGpuPtr[i].normal_y = normalsGpuPtr[i].y;
        // pcGpuPtr[i].normal_z = normalsGpuPtr[i].z;
    }

    void checkCudaError(hipError_t err) {
        if (err == hipSuccess) return;

        //RCLCPP_ERROR_STREAM(rclcpp::get_logger("cuda_error"), "CUDA error: " << hipGetErrorString(err));
        throw std::runtime_error("CUDA error");
    }

    /**
     * Fills a PointCloud2 message residing on the CPU from two GPU buffers (one for XYZ and one for BGRA).
     *
     * @param xyzGpu    XYZ buffer on the GPU
     * @param bgraGpu   BGRA buffer on the GPU
     * @param pcGpu     Point cloud buffer on the GPU (@see Point)
     * @param msg       Point cloud message with buffer on the CPU
     */
    void fillPointCloudMessageFromGpu(sl::Mat& xyzGpu, sl::Mat& bgraGpu, sl::Mat& normalsGpu, PointCloudGpu& pcGpu, sensor_msgs::msg::PointCloud2::UniquePtr const& msg) {
        assert(bgraGpu.getWidth() >= xyzGpu.getWidth());
        assert(bgraGpu.getHeight() >= xyzGpu.getHeight());
        assert(bgraGpu.getChannels() == 4);
        assert(xyzGpu.getChannels() == 4); // Last channel is unused
        assert(msg);

        auto* bgraGpuPtr = bgraGpu.getPtr<sl::uchar4>(sl::MEM::GPU);
        auto* xyzGpuPtr = xyzGpu.getPtr<sl::float4>(sl::MEM::GPU);
        auto* normalsGpuPtr = normalsGpu.getPtr<sl::float4>(sl::MEM::GPU);
        msg->is_bigendian = __BYTE_ORDER__ == __ORDER_BIG_ENDIAN__;
        msg->is_dense = true;
        msg->height = bgraGpu.getHeight();
        msg->width = bgraGpu.getWidth();
        fillPointCloudMessageHeader(msg);
        std::size_t size = msg->width * msg->height;

        pcGpu.resize(size);
        Point* pcGpuPtr = pcGpu.data().get();
        dim3 threadsPerBlock{BLOCK_SIZE};
        dim3 numBlocks{static_cast<uint>(std::ceil(static_cast<float>(size) / BLOCK_SIZE))};
        fillPointCloudMessageKernel<<<numBlocks, threadsPerBlock>>>(xyzGpuPtr, bgraGpuPtr, normalsGpuPtr, pcGpuPtr, size);
        checkCudaError(hipPeekAtLastError());
        checkCudaError(hipMemcpy(msg->data.data(), pcGpuPtr, size * sizeof(Point), hipMemcpyDeviceToHost));
    }


} // namespace mrover
